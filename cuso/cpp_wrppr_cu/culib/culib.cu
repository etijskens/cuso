#include "hip/hip_runtime.h"
#include "culib.hpp"


// Kernel function to add the elements of two arrays
__global__
void add(int n, double *x, double *y, double*z)
{
  for (int i = 0; i < n; i++)
    z[i] = x[i] + y[i];
}


void 
culib_add
    ( double const* a
    , double const* b
    , double* c // a+b 
    , size_t n
    )
{
  double *cu_a, *cu_b, *cu_c;

  // Allocate Unified Memory – accessible from CPU or GPU
  hipMallocManaged(&cu_a, n*sizeof(double));
  hipMallocManaged(&cu_b, n*sizeof(double));
  hipMallocManaged(&cu_c, n*sizeof(double));

  // initialize a and b arrays on the host
    for( size_t i = 0; i < n; ++i ) {
        cu_a[i] = a[i];
        cu_b[i] = b[i];
    }

 // Run kernel on 1M elements on the GPU
    add<<<1, 1>>>(n, cu_a, cu_b, cu_c);

 // Wait for GPU to finish before accessing on host
    hipDeviceSynchronize();

    for( size_t i = 0; i < n; ++i ) {
        c[i] = cu_c[i];
    }
}